#include "hip/hip_runtime.h"
#include "opencv2/highgui/highgui.hpp"
#include <iostream>
#include <stdio.h>

#include <hip/hip_runtime.h>

using namespace cv;
using namespace std;


Mat img;
Mat new_img;
int radio, NUM_THREADS;

// CUDA API error checking macro
static void handleError( hipError_t err, const char *file,  int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),  file, line );
        exit( EXIT_FAILURE );
    }
}
#define cudaCheck( err ) (handleError( err, __FILE__, __LINE__ ))

//Funcion que se realiza en loshilos para generar el efecto borroso
__global__ void blur(int *r_in, int *r_out,int *g_in, int *g_out,int *b_in, int *b_out, int radio, int numthreads, int numblocks, int largo) {
   	int gindex = threadIdx.x + (blockIdx.x * blockDim.x);
		int aux = largo/(numthreads*numblocks)+1;
		for (int i=gindex*aux; i<(gindex+1)*(aux); i++){ 
		int aux_r = 0, aux_g = 0, aux_b = 0, count=0;

	 			for (int a=0; a<=radio; a++){
	 				if(!(i-a<0)){
	 					aux_r+= r_in[i-a];
	 					aux_g+= g_in[i-a];
	 					aux_b+= b_in[i-a];
						count++;
	 				}
	 				if(i+a<largo){
	 					aux_r+= r_in[i+a];
	 					aux_g+= g_in[i+a];
	 					aux_b+= b_in[i+a];
						count++;
	 				}
	 			}
				

			//promedio de pixeles sumados
			aux_r = int(aux_r/(count));
			aux_g = int(aux_g/(count));
			aux_b = int(aux_b/(count));

			//guardado del valor a enviar al host del nuevo valor del pixel

			r_out[i] = aux_r;
			g_out[i] = aux_g;
			b_out[i] = aux_b;
		}
}

//Funcion main
int main( int argc, char** argv )
{
  NUM_THREADS = atoi(argv[3]);
  int num_blocks = atoi(argv[4]);
  int  j, k;

  //Abrir las imagenes y guardarlas en memoria
  img = imread(argv[1], CV_LOAD_IMAGE_UNCHANGED);
  new_img = imread(argv[1], CV_LOAD_IMAGE_UNCHANGED);

  if (img.empty()){
      cout << "Error : Image cannot be loaded..!!" << endl;
      return -1;
  }

	int h_r_in[img.cols],h_r_out[img.cols],h_g_in[img.cols],h_g_out[img.cols],h_b_in[img.cols],h_b_out[img.cols];
 	//variables de device
  int *d_r_in, *d_r_out,*d_g_in, *d_g_out,*d_b_in, *d_b_out;
  radio=atoi(argv[2]);
	int largo=img.cols;
  //Reserva de recursos en device
  hipMalloc( (void **) &d_r_in, img.cols * sizeof(int));
	hipMalloc( (void **) &d_r_out, img.cols * sizeof(int));
	hipMalloc( (void **) &d_g_in, img.cols * sizeof(int));
	hipMalloc( (void **) &d_g_out, img.cols * sizeof(int));
	hipMalloc( (void **) &d_b_in, img.cols * sizeof(int));
	hipMalloc( (void **) &d_b_out, img.cols * sizeof(int));

  //k recorre fila por fila
  for(j=0;j<img.rows;j++){
      //asigna los valores de la fila actual en el host
      for(k=0;k<img.cols;k++){
        h_r_in[k] = int(img.at<Vec3b>(j,k)[0]);
				h_g_in[k] = int(img.at<Vec3b>(j,k)[1]);
				h_b_in[k] = int(img.at<Vec3b>(j,k)[2]);
			}

      //envia los valores de la fila actual del host al device
      cudaCheck( hipMemcpy( d_r_in, h_r_in, img.cols * sizeof(int), hipMemcpyHostToDevice));
			cudaCheck( hipMemcpy( d_g_in, h_g_in, img.cols * sizeof(int), hipMemcpyHostToDevice));
			cudaCheck( hipMemcpy( d_b_in, h_b_in, img.cols * sizeof(int), hipMemcpyHostToDevice));
      //ejecuta el stencil
      blur<<<num_blocks,NUM_THREADS>>> (d_r_in,d_r_out,d_g_in,d_g_out,d_b_in,d_b_out, radio, NUM_THREADS, num_blocks, largo);

      //guarda en el host los valores generados por el stencil
      hipMemcpy( h_r_out, d_r_out, img.cols * sizeof(int), hipMemcpyDeviceToHost);
      hipMemcpy( h_g_out, d_g_out, img.cols * sizeof(int), hipMemcpyDeviceToHost);
      hipMemcpy( h_b_out, d_b_out, img.cols * sizeof(int), hipMemcpyDeviceToHost);
      //recorre la fila actual y le asigna los nuevos valores rgb
      for(k=0;k<img.cols;k++){
        new_img.at<Vec3b>(j,k)[0] = h_r_out[k];
        new_img.at<Vec3b>(j,k)[1] = h_g_out[k];
        new_img.at<Vec3b>(j,k)[2] = h_b_out[k];
			}
  }
	

  string name = "modificada_";
  name.append("kernel_");
  name.append(argv[2]);
	name.append("_");
  name.append(argv[1]);
  //Guardar la imagen
  imwrite(name, new_img);
  return 0;
}


